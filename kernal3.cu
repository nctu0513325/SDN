#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__device__ int mandel(float c_re, float c_im, int count) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;
        float new_re = (z_re * z_re) - (z_im * z_im);
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandel_kernel(float lower_x, float lower_y, float step_x, float step_y, int *img, size_t pitch, int res_x, int res_y, int max_iterations) {
    __shared__ float shared_lower_x;
    __shared__ float shared_lower_y;
    __shared__ float shared_step_x;
    __shared__ float shared_step_y;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_lower_x = lower_x;
        shared_lower_y = lower_y;
        shared_step_x = step_x;
        shared_step_y = step_y;
    }
    __syncthreads();

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int groupSize = 4; // Example group size
    for (int i = 0; i < groupSize; ++i) {
        int pixelX = thisX * groupSize + i;
        if (pixelX < res_x && thisY < res_y) {
            float x = shared_lower_x + pixelX * shared_step_x;
            float y = shared_lower_y + thisY * shared_step_y;
            int *row = (int*)((char*)img + thisY * pitch);
            row[pixelX] = mandel(x, y, max_iterations);
        }
    }
}

void host_fe(float upper_x, float upper_y, float lower_x, float lower_y, int *img, int res_x, int res_y, int max_iterations) {
    float step_x = (upper_x - lower_x) / (float)res_x;
    float step_y = (upper_y - lower_y) / (float)res_y;

    int *host_img;
    hipHostAlloc(&host_img, res_x * res_y * sizeof(int), hipHostMallocDefault);

    int *device_img;
    size_t pitch;
    hipMallocPitch(&device_img, &pitch, res_x * sizeof(int), res_y);

    dim3 threadsPerBlock(16, 16);  // 使用16x16的线程块大小
    dim3 numBlocks((res_x + threadsPerBlock.x * 4 - 1) / (threadsPerBlock.x * 4), (res_y + threadsPerBlock.y - 1) / threadsPerBlock.y);

    mandel_kernel<<<numBlocks, threadsPerBlock>>>(lower_x, lower_y, step_x, step_y, device_img, pitch, res_x, res_y, max_iterations);

    hipMemcpy2D(host_img, res_x * sizeof(int), device_img, pitch, res_x * sizeof(int), res_y, hipMemcpyDeviceToHost);

    // Copy host_img to img
    for (int i = 0; i < res_x * res_y; ++i) {
        img[i] = host_img[i];
    }

    hipHostFree(host_img);
    hipFree(device_img);
}
